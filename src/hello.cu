
#include <hip/hip_runtime.h>
const int blocksize = 16;

__global__ void hello(char *a, int *b)
{
    a[threadIdx.x] += b[threadIdx.x];
}


void run(char * string, int * vector, int csize, int isize)
{
    char *ad;
    int *bd;


    hipMalloc((void**)&ad, csize);
    hipMalloc((void**)&bd, isize);
    hipMemcpy(ad, string, csize, hipMemcpyHostToDevice);
    hipMemcpy(bd, vector, isize, hipMemcpyHostToDevice);

    dim3 dimBlock(blocksize, 1);
    dim3 dimGrid(1, 1);
    hello<<<dimGrid, dimBlock>>>(ad, bd);
    hipMemcpy(string, ad, csize, hipMemcpyDeviceToHost);
    hipFree(ad);
    hipFree(bd);
}
